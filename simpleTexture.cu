#include "hip/hip_runtime.h"
/*
todo:
1. Dispersion : code source sur la page du prof
2. Median filter
2. Sobel dans une copie
3. Multiplication du Sobel et du median
*/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>

// Includes CUDA
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

//#include opencv
#include <opencv2/opencv.hpp>

__device__ void triArray(unsigned char *a, const int size){
    float tmp;
    for(int i = 0; i < size; i ++){
        for(int j = 0; j < size-1; j++){
            if(a[j] > a[j+1]){
                tmp = a[j];
                a[j] = a[j+1];
                a[j+1] = tmp;
            }
        }
    }
}

__device__ void medianFilter(unsigned char*input, unsigned char *outputData, unsigned char*devMed, int window, int width,int height){
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int id = y * width + x, idx3 = 3*id;
    unsigned int w2 = window*window, arrayId = 3*w2*id;
    int w_div2 = window/2;

    //to find the median -> tri à bulle + valeur du milieu
    if(x > w_div2 && x < width-w_div2 && y > w_div2 && y < height - w_div2){
        int index = 0;
        for(int i = -w_div2; i <= w_div2; i++){
            for(int j = -w_div2; j <= w_div2; j++){
                devMed[arrayId + index] = input[3 * ((y+j)*width+(x+i)) + 0];
                devMed[arrayId + w2 + index] = input[3 * ((y+j)*width+(x+i)) + 1];
                devMed[arrayId + 2*w2 + index] = input[3 * ((y+j)*width+(x+i)) + 2];
                index++;
            }
        }
        triArray(&devMed[arrayId], w2);triArray(&devMed[arrayId + w2], w2);triArray(&devMed[arrayId + 2*w2], w2);
        outputData[idx3 + 0] = devMed[arrayId + (w2/2)];//apply median
        outputData[idx3 + 1] = devMed[arrayId + w2 + (w2/2)];
        outputData[idx3 + 2] = devMed[arrayId + 2*w2 + (w2/2)];
    }else{//copy
        outputData[idx3 + 0] =  input[idx3 + 0];
        outputData[idx3 + 1] =  input[idx3 + 0];
        outputData[idx3 + 2] =  input[idx3 + 0];
    }
}

__device__ void sobelFilter(unsigned char *input, unsigned char *outputData, int width, int height){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int id = y * width + x;

    if(x > 0 && x < width-1 && y > 0 && y < height -1){
        char sobel_x[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
        char sobel_y[9] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};
        float b_pixel[2] = {0, 0}, g_pixel[2] = {0, 0}, r_pixel[2] = {0, 0};
        int index = 0;
        for(int i = -1; i < 2; i++){
            for(int j = -1; j < 2; j++){
                b_pixel[0] += input[3*((y+j)*width + (x+i))] * sobel_x[index];
                b_pixel[1] += input[3*((y+j)*width + (x+i))] * sobel_y[index];
                g_pixel[0] += input[3*((y+j)*width + (x+i))] * sobel_x[index];
                g_pixel[1] += input[3*((y+j)*width + (x+i))] * sobel_y[index];
                r_pixel[0] += input[3*((y+j)*width + (x+i))] * sobel_x[index];
                r_pixel[1] += input[3*((y+j)*width + (x+i))] * sobel_y[index];
                index++;
            }
        }

        float b = sqrt((b_pixel[0]*b_pixel[0])+(b_pixel[1]*b_pixel[1]));
        float g = sqrt((g_pixel[0]*g_pixel[0])+(g_pixel[1]*g_pixel[1]));
        float r = sqrt((r_pixel[0]*r_pixel[0])+(r_pixel[1]*r_pixel[1]));

        outputData[3 * id + 0] = (b>128)?255:b;
        outputData[3 * id + 1] = (g>128)?255:g;
        outputData[3 * id + 2] = (r>128)?255:r;
    }
}

__device__ void multiply(unsigned char*input_1, unsigned char*output, int width, int height){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int id = y * width + x;

    output[3 * id + 0] *= input_1[3 * id + 0] / 255.0f;
    output[3 * id + 1] *= input_1[3 * id + 1] / 255.0f;
    output[3 * id + 2] *= input_1[3 * id + 2] / 255.0f;
}

__device__ void dispersionFilter(unsigned char*input, unsigned char *outputData,char*commutation_array, int window, int width, int height){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int id = y * width + x, w_div2 = window/2, arrayId = 3*id;

    if(x > w_div2 && y > w_div2 && x < width-w_div2 && y < height-w_div2){
        unsigned int index = 3 * ((y+commutation_array[id+1])*width + x + commutation_array[id]);
        outputData[arrayId+0] = input[index+0];
        outputData[arrayId+1] = input[index+1];
        outputData[arrayId+2] = input[index+2];
    }else outputData[arrayId] = input[arrayId];
}

__global__ void applyFilters(unsigned char *input, unsigned char *outputData, char*commutation_array, unsigned char*devMed, int window,int width, int height){
    dispersionFilter(input, outputData,commutation_array, window, width, height);
    __syncthreads();
    medianFilter(outputData, input, devMed, window, width, height);
    __syncthreads();
    sobelFilter(input, outputData, width, height);//result of sobel in input
    __syncthreads();
    multiply(input, outputData, width, height);//multiply result of sobel with result of median*/
}

void getCommutationArray(char* arr, int size, unsigned int window){for(int i = 0; i < size; i++)arr[i] = (char)rand()%window - (window/2);}

bool processEvent(bool&continuer, int&window){
    bool result = true;
    char carac = cv::waitKey(33);
    switch(carac){
        case 32://spacebar
            continuer = false;
        break;
        case 82://up
            window = (window < 11) ? window+2 : 11;
            printf("window size = %dx%d \n",window, window);
        break;
        case 84://down
            window = (window > 3) ? window - 2 : window;
            printf("window size = %dx%d \n",window, window);
        break;
        default:result = false;break;
    }
    return result;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv){
    findCudaDevice(argc, (const char **) argv);
    srand(time(NULL));
    bool continuer = true, config = true;
    int width = 0, height = 0, window = 5;
    cv::VideoCapture cam(0); cv::Mat img;
    unsigned char*data = NULL, *devData = NULL, *devBuffer = NULL, *devMed = NULL;
    cam >> img; width = img.cols; height = img.rows; data = (unsigned char*)img.data;//getting size of the frame

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);
    
    checkCudaErrors(hipMalloc((void**) &devData, 3*(width*height) * sizeof(char)));
    checkCudaErrors(hipMalloc((void**) &devBuffer, 3*(width*height) * sizeof(char)));
    checkCudaErrors(hipMalloc((void**) &devMed, 3*window*window*(width*height) * sizeof(char)));
    char*com = (char*)malloc((width*height+1)*sizeof(char)), *devCom;
    checkCudaErrors(hipMalloc((void**) &devCom, (width*height+1) * sizeof(char)));

    while(continuer){
        if(config){
            config = false;
            getCommutationArray(com, width*height+1, window);
            checkCudaErrors(hipMemcpy(devCom, com, (width*height+1)*sizeof(char), hipMemcpyHostToDevice));
            checkCudaErrors(hipFree(devMed));
            checkCudaErrors(hipMalloc((void**) &devMed, 3*window*window*(width*height) * sizeof(char)));
        }
        cam >> img; width = img.cols; height = img.rows; data = (unsigned char*)img.data;
        checkCudaErrors(hipMemcpy(devData, data, 3*width*height*sizeof(char), hipMemcpyHostToDevice));
        //begin main

        StopWatchInterface *timer = NULL;
        sdkCreateTimer(&timer);
        sdkStartTimer(&timer);
        applyFilters<<<dimGrid, dimBlock, 0>>>(devData, devBuffer, devCom, devMed, window, width, height);
        checkCudaErrors(hipMemcpy(data, devData, 3*width*height*sizeof(char), hipMemcpyDeviceToHost));
        sdkStopTimer(&timer);
        printf("Processing time: %f (ms)\r\n", sdkGetTimerValue(&timer));

        //end main

        cv::imshow("test", img);
        config = processEvent(continuer, window);
    }

    cam.release();
    free(com);
    checkCudaErrors(hipFree(devData));
    checkCudaErrors(hipFree(devBuffer));
    checkCudaErrors(hipFree(devMed));
    checkCudaErrors(hipFree(devCom));
    return 0;
}
