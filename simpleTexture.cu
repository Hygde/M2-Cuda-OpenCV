#include "hip/hip_runtime.h"
/*
todo:
1. Dispersion : code source sur la page du prof
2. Median filter
2. Sobel dans une copie
3. Multiplication du Sobel et du median
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// Includes CUDA
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

//#include opencv
#include <opencv2/opencv.hpp>
//#include <opencv2/gpu/gpu.hpp>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#define MAX_EPSILON_ERROR 5e-3f

// Define the files that are to be save and the reference images for validation
const char *imageFilename = "lena_bw.pgm";
const char *refFilename   = "ref_rotated.pgm";

const char *sampleName = "simpleTexture";

////////////////////////////////////////////////////////////////////////////////
// Texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;

// Auto-Verification Code
bool testResult = true;

__device__ void triArray(float *a, const int size){
    float tmp;
    for(int i = 0; i < size; i ++){
        for(int j = 0; j < size-1; j++){
            if(a[j] > a[j+1]){
                tmp = a[j];
                a[j] = a[j+1];
                a[j+1] = tmp;
            }
        }
    }
}

__device__ void medianFilter(float *outputData,int width,int height){
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    //float u = x / (width - 0.0f);
    //float v = y / (height - 0.0f);

    //to find the median -> tri à bulle + valeur du milieu
    float values[9];
    if(x > 0 && x < width-1 && y > 0 && y < height -1){
        int index = 0;
        for(int i = -1; i < 2; i++){
            for(int j = -1; j < 2; j++){
                values[index] = outputData[(y+j)*width + (x+i)];
                index++;
            }
        }
    }

    triArray(values, 9);

    // read from texture and write to global memory
    outputData[y*width + x] = values[4];
}

__device__ void sobelFilter(float *input, float*outputData, int width, int height){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x > 0 && x < width-1 && y > 0 && y < height -1){
        char sobel_x[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
        char sobel_y[9] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};
        float pixel_x = 0.0, pixel_y = 0.0;
        int index = 0;
        for(int i = -1; i < 2; i++){
            for(int j = -1; j < 2; j++){
                pixel_x += input[(y+j)*width + (x+i)] * sobel_x[index];
                pixel_y += input[(y+j)*width + (x+i)] * sobel_y[index];
                index++;
            }
        }
        outputData[y*width + x] = sqrt( (pixel_x*pixel_x) + (pixel_y*pixel_y)); 
    }
}

__device__ void multiply(float*input_1, float*output, int width, int height){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    output[y*width + x] *= input_1[y*width + x];
}

__device__ void dispersionFilter(float *outputData, int width, int height){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    
    float xstep = 1.0f / width, ystep = 1.0f / height;
    float u = x * xstep, v = y * ystep;

    if(x > 0 && y > 0){
        outputData[y*width + x] = tex2D(tex, u+xstep*(-1), v+ystep*(-1));
        outputData[(y-1)*width + x-1] = tex2D(tex, u+xstep, v+ystep);
    }else{
        outputData[y*width + x] = tex2D(tex, u+xstep*2, v+ystep*2);
        outputData[(y+1)*width + x+1] = tex2D(tex, u+xstep, v+ystep);
    }
}

__global__ void applyFilters(float *input, float *outputData, int width, int height){
    dispersionFilter(outputData, width, height);
    __syncthreads();
    medianFilter(outputData, width, height);
    __syncthreads();
    sobelFilter(outputData, input, width, height);//result of sobel in input
    __syncthreads();
    multiply(input, outputData, width, height);//multiply result of sobel with result of median
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("%s starting...\n", sampleName);
    // Process command-line arguments
    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **) argv, "input"))
        {
            getCmdLineArgumentString(argc,(const char **) argv,"input",(char **) &imageFilename);

            if (checkCmdLineFlag(argc, (const char **) argv, "reference"))getCmdLineArgumentString(argc,(const char **) argv,"reference",(char **) &refFilename);
            else{
                printf("-input flag should be used with -reference flag");
                exit(EXIT_FAILURE);
            }
        }
        else if (checkCmdLineFlag(argc, (const char **) argv, "reference")){
            printf("-reference flag should be used with -input flag");
            exit(EXIT_FAILURE);
        }
    }

    runTest(argc, argv);

    printf("%s completed, returned %s\n",sampleName,testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv){
    int devID = findCudaDevice(argc, (const char **) argv);

    // load image from disk
    float *hData = NULL;
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageFilename, argv[0]);

    if (imagePath == NULL){
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }

    sdkLoadPGM(imagePath, &hData, &width, &height);

    unsigned int size = width * height * sizeof(float);
    printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

    //Load reference image from image (output)
    float *hDataRef = (float *) malloc(size);
    char *refPath = sdkFindFilePath(refFilename, argv[0]);

    // Allocate device memory for result
    float *input_1 = NULL, *output = NULL;
    checkCudaErrors(hipMalloc((void **) &input_1, size));
    checkCudaErrors(hipMalloc((void **) &output, size));

    // Allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray *cuArray;
    checkCudaErrors(hipMallocArray(&cuArray,&channelDesc,width,height));
    checkCudaErrors(hipMemcpyToArray(cuArray,0,0,hData,size,hipMemcpyHostToDevice));

    // Set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModePoint;//hipFilterModeLinear;
    tex.normalized = true;// access with normalized texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(tex, cuArray, channelDesc));

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    checkCudaErrors(hipDeviceSynchronize());
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Allocate mem for the result on host side
    float *hOutputData = (float *) malloc(size);

    // Execute the kernel
    applyFilters<<<dimGrid, dimBlock, 0>>>(input_1, output, width, height);

    // Check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    printf("%.2f Mpixels/sec\n",(width *height / (sdkGetTimerValue(&timer) / 1000.0f)) / 1e6);
    sdkDeleteTimer(&timer);

    // copy result from device to host
    checkCudaErrors(hipMemcpy(hOutputData,output, size,hipMemcpyDeviceToHost));

    // Write result to file
    char outputFilename[1024];
    strcpy(outputFilename, imagePath);
    strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");
    sdkSavePGM(outputFilename, hOutputData, width, height);
    printf("Wrote '%s'\n", outputFilename);

    checkCudaErrors(hipFree(input_1));
    checkCudaErrors(hipFree(output));

    checkCudaErrors(hipFreeArray(cuArray));
    free(imagePath);
    free(hOutputData);
}
