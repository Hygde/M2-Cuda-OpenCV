#include "hip/hip_runtime.h"
/*
todo:
1. Dispersion : code source sur la page du prof
2. Median filter
2. Sobel dans une copie
3. Multiplication du Sobel et du median
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// Includes CUDA
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

//#include opencv
#include <opencv2/opencv.hpp>
//#include <opencv2/gpu/gpu.hpp>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#define MAX_EPSILON_ERROR 5e-3f

// Define the files that are to be save and the reference images for validation
const char *imageFilename = "lena_bw.pgm";
const char *refFilename   = "ref_rotated.pgm";

const char *sampleName = "simpleTexture";

////////////////////////////////////////////////////////////////////////////////
// Texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;

// Auto-Verification Code
bool testResult = true;

__device__ void triArray(float *a, const int size){
    float tmp;
    for(int i = 0; i < size; i ++){
        for(int j = 0; j < size-1; j++){
            if(a[j] > a[j+1]){
                tmp = a[j];
                a[j] = a[j+1];
                a[j+1] = tmp;
            }
        }
    }
}

__device__ void medianFilter(float *outputData,int width,int height){
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    //float u = x / (width - 0.0f);
    //float v = y / (height - 0.0f);

    //to find the median -> tri à bulle + valeur du milieu
    float values[9];
    if(x > 0 && x < width-1 && y > 0 && y < height -1){
        int index = 0;
        for(int i = -1; i < 2; i++){
            for(int j = -1; j < 2; j++){
                values[index] = outputData[(y+j)*width + (x+i)];
                index++;
            }
        }
    }

    triArray(values, 9);

    // read from texture and write to global memory
    outputData[y*width + x] = values[4];
}

__device__ void dispersionFilter(float *outputData, int width, int height){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    int acom[3] = {-1,0,1};

    float xstep = 1.0f / width, ystep = 1.0f / height;
    float u = x * xstep, v = y * ystep;

    outputData[y*width + x] = tex2D(tex, u+xstep*acom[threadIdx.x%3], v+ystep*acom[threadIdx.y%3]);
}

__global__ void applyFilters(float *outputData, int width, int height){
    dispersionFilter(outputData, width, height);
    __syncthreads();
    medianFilter(outputData, width, height);
    __syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("%s starting...\n", sampleName);
    // Process command-line arguments
    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **) argv, "input"))
        {
            getCmdLineArgumentString(argc,(const char **) argv,"input",(char **) &imageFilename);

            if (checkCmdLineFlag(argc, (const char **) argv, "reference"))getCmdLineArgumentString(argc,(const char **) argv,"reference",(char **) &refFilename);
            else{
                printf("-input flag should be used with -reference flag");
                exit(EXIT_FAILURE);
            }
        }
        else if (checkCmdLineFlag(argc, (const char **) argv, "reference")){
            printf("-reference flag should be used with -input flag");
            exit(EXIT_FAILURE);
        }
    }

    runTest(argc, argv);

    printf("%s completed, returned %s\n",sampleName,testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv){
    int devID = findCudaDevice(argc, (const char **) argv);

    // load image from disk
    float *hData = NULL;
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageFilename, argv[0]);

    if (imagePath == NULL){
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }

    sdkLoadPGM(imagePath, &hData, &width, &height);

    unsigned int size = width * height * sizeof(float);
    printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

    //Load reference image from image (output)
    float *hDataRef = (float *) malloc(size);
    char *refPath = sdkFindFilePath(refFilename, argv[0]);

    if (refPath == NULL){
        printf("Unable to find reference image file: %s\n", refFilename);
        exit(EXIT_FAILURE);
    }

    sdkLoadPGM(refPath, &hDataRef, &width, &height);

    // Allocate device memory for result
    float *dData = NULL, *iData = NULL;
    checkCudaErrors(hipMalloc((void **) &dData, size));
    checkCudaErrors(hipMalloc((void **) &iData, size));

    // Allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray *cuArray;
    checkCudaErrors(hipMallocArray(&cuArray,&channelDesc,width,height));
    checkCudaErrors(hipMemcpyToArray(cuArray,0,0,hData,size,hipMemcpyHostToDevice));

    // Set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModePoint;//hipFilterModeLinear;
    tex.normalized = true;// access with normalized texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(tex, cuArray, channelDesc));

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    checkCudaErrors(hipDeviceSynchronize());
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Allocate mem for the result on host side
    float *hOutputData = (float *) malloc(size);

    // Execute the kernel
    applyFilters<<<dimGrid, dimBlock, 0>>>(dData, width, height);

    // Check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    printf("%.2f Mpixels/sec\n",(width *height / (sdkGetTimerValue(&timer) / 1000.0f)) / 1e6);
    sdkDeleteTimer(&timer);

    // copy result from device to host
    checkCudaErrors(hipMemcpy(hOutputData,dData,size,hipMemcpyDeviceToHost));

    // Write result to file
    char outputFilename[1024];
    strcpy(outputFilename, imagePath);
    strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");
    sdkSavePGM(outputFilename, hOutputData, width, height);
    printf("Wrote '%s'\n", outputFilename);

    checkCudaErrors(hipFree(dData));
    checkCudaErrors(hipFree(iData));
    checkCudaErrors(hipFreeArray(cuArray));
    free(imagePath);
    free(refPath);
    free(hOutputData);
}
